/*
 * Title: CS6023, GPU Programming, Jan-May 2023, Assignment-1
 * Description: Computation of a matrix C = Kronecker_prod(A, B.T)
 *              where A and B are matrices of dimension (m, n) and
 *              the output is of the dimension (m * n, m * n). 
 * Note: All lines marked in --> should be replaced with code. 
 */

#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
using namespace std;

ofstream outfile; // The handle for printing the output

__global__ void per_row_AB_kernel(long int *A, long int *B, long int *C,long int m, long int n){
    long long int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < m * m)
    {
        long int i = row / m;
        long int j = row % m;
        for (long int k = 0; k < n; k++)
        {
            for (long int l = 0; l < n; l++)
            {
                C[(n*i+l)*m*n+m*k+j] = A[i * n + k] * B[j * n + l];
            }
        }
    }
}

__global__ void per_column_AB_kernel(long int *A, long int *B, long int *C,long int m, long int n){
    long int col = blockIdx.x * blockDim.x *blockDim.y + threadIdx.x*blockDim.y+threadIdx.y;
    if (col < n * n)
    {
        long int i = col / n;
        long int j = col % n;
        for (long int k = 0; k < m; k++)
        {
            for (long int l = 0; l < m; l++)
            {
                C[(n*k+j)*m*n+m*i+l] = A[k * n + i] * B[l * n + j];
            }
        }
    }
}

__global__ void per_element_kernel(long int *A, long int *B, long int *C,long int m, long int n){
    
    long long int id = blockIdx.y * gridDim.x * blockDim.y * blockDim.x + blockIdx.x * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;
    if(id <m*n*m*n)
    {
        long int i = (id / (m*n*n));
        long int j = ((id % (m*n)) / m);
        long int k = id % m;
        long int l = (id % (m*n*n) / (m*n));
        C[id] = A[i * n + j] * B[k * n + l];
    }
}
/**
 * Prints any 1D array in the form of a matrix
 **/
void printMatrix(long int *arr, long int rows, long int cols, char* filename){
    outfile.open(filename);
    for(long int i = 0; i < rows; i++){
        for(long int j = 0; j < cols; j++){
            outfile<<arr[i * cols + j]<<" ";
        }
        outfile<<"\n";
    }
    outfile.close();
}

/**
 * Timing functions taken from the matrix multiplication source code
 * rtclock - Returns the time of the day 
 * printtime - Prints the time taken for computation 
 **/
double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main(int argc,char **argv){
    // Variable declarations
    long int m,n;	
    cin>>m>>n;	

    // Host_arrays 
    long int *h_a,*h_b,*h_c;

    // Device arrays 
    long int *d_a,*d_b,*d_c;
	
    // Allocating space for the host_arrays 
    h_a = (long int *) malloc(m * n * sizeof(long int));
    h_b = (long int *) malloc(m * n * sizeof(long int));	
    h_c = (long int *) malloc(m * m * n * n * sizeof(long int));	

    // Allocating memory for the device arrays 
    hipMalloc(&d_a,m*n*sizeof(long int));
    hipMalloc(&d_b,m*n*sizeof(long int));
    hipMalloc(&d_c,m*m*n*n*sizeof(long int));

    // Read the input matrix A 
    for(long int i = 0; i < m * n; i++) {
        cin>>h_a[i];
    }

    //Read the input matrix B 
    for(long int i = 0; i < m * n; i++) {
        cin>>h_b[i];
    }

    // Transfer the input host arrays to the device 
    hipMemcpy(d_a,h_a,m*n*sizeof(long int),hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,m*n*sizeof(long int),hipMemcpyHostToDevice);

    long int gridDimx, gridDimy;
    
    // Launch the kernels
    /**
     * Kernel 1 - per_row_AB_kernel
     * To be launched with 1D grid, 1D block
     * Each thread should process a complete row of A, B
     **/

    gridDimx = ceil(float(m * m) / 1024);
    dim3 gridrow(gridDimx,1,1);
    dim3 blockrow(1024, 1, 1);  
    double starttime = rtclock();  

    per_row_AB_kernel<<<gridrow,blockrow>>>(d_a,d_b,d_c,m,n);
    hipDeviceSynchronize();                                                           

    double endtime = rtclock(); 
	printtime("GPU Kernel-1 time: ", starttime, endtime);  

    hipMemcpy(h_c,d_c,m*m*n*n*sizeof(long int),hipMemcpyDeviceToHost); 

    printMatrix(h_c, m * n, m * n,"kernel1.txt");
    hipMemset(d_c, 0, m * n * m * n * sizeof(long int));

    /**
     * Kernel 2 - per_column_AB_kernel
     * To be launched with 1D grid, 2D block
     * Each thread should process a complete column of  A, B
     **/
    
    gridDimx = ceil(float(n * n) / 1024);
    dim3 gridcol(gridDimx,1,1);
    dim3 blockcol(32,32, 1);
    starttime = rtclock(); 

    per_column_AB_kernel<<<gridcol,blockcol>>>(d_a,d_b,d_c,m,n);
    hipDeviceSynchronize(); 

    endtime = rtclock(); 
  	printtime("GPU Kernel-2 time: ", starttime, endtime);  

    hipMemcpy(h_c,d_c,m*n*m*n*sizeof(long int),hipMemcpyDeviceToHost); 
    
    printMatrix(h_c, m * n, m * n,"kernel2.txt");
    hipMemset(d_c, 0, m * n * m * n * sizeof(long int));

    /**
     * Kernel 3 - per_element_kernel
     * To be launched with 2D grid, 2D block
     * Each thread should process one element of the output 
     **/
    gridDimx = ceil(float(n * n) / 16);
    gridDimy = ceil(float(m * m) / 64);
    dim3 grid(gridDimx,gridDimy,1);
    dim3 block(64,16,1);

    starttime = rtclock();  

    per_element_kernel<<<grid,block>>>(d_a,d_b,d_c,m,n);
    hipDeviceSynchronize();                                                              

    endtime = rtclock();  
	printtime("GPU Kernel-3 time: ", starttime, endtime);  

    hipMemcpy(h_c,d_c,m*n*m*n*sizeof(long int),hipMemcpyDeviceToHost); 
    
    printMatrix(h_c, m * n, m * n,"kernel3.txt");

    return 0;
}